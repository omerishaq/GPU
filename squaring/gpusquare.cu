
#include <hip/hip_runtime.h>
/* Kernel for vector squaring */

__global__ void gpusquare(float in[], float out[], int n) 
{
   
   int i = blockDim.x * blockIdx.x + threadIdx.x;

   if (i < n)
	{ 
	out[i] = in[i] * in[i];
	}

}  